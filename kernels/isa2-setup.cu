#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cuh"

#ifdef ISA2_0_1_2
#include "isa2-0-1-2.h"
#elif defined(ISA2_01_2)
#include "isa2-01-2.h"
#elif defined(ISA2_0_12)
#include "isa2-0-12.h"
#elif defined(ISA2_012)
#include "isa2-012.h"
#else
#include "isa2-0-1-2.h" // default to all different threadblocks
#endif

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params) {

  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

#ifdef ACQUIRE
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_acquire;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
#elif defined(RELEASE)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_release;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
#elif defined(ACQUIRE_RELEASE)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_acquire;
    cuda::memory_order thread_1_store = cuda::memory_order_release;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
#elif defined(RELAXED)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
#else
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed; // default to all relaxed
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
#endif

    // defined for different distributions of threads across threadblocks
    DEFINE_IDS();

    uint x_0 = (wg_offset + id_0) * kernel_params->mem_stride * 3;
    uint y_0 = (wg_offset + permute_id(id_0, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 3 + kernel_params->mem_offset;
    uint permute_id_1 = permute_id(id_1, kernel_params->permute_location, total_ids);
    uint y_1 = (wg_offset + permute_id_1) * kernel_params->mem_stride * 3 + kernel_params->mem_offset;
    uint z_1 = (wg_offset + permute_id(permute_id_1, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 3 + 2 * kernel_params->mem_offset;
    uint x_2 = (wg_offset + id_2) * kernel_params->mem_stride * 3;
    uint permute_id_2 = permute_id(id_2, kernel_params->permute_location, total_ids);
    uint z_2 = (wg_offset + permute_id(permute_id_2, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 3 + 2 * kernel_params->mem_offset;

    if (kernel_params->pre_stress) {
      do_stress(scratchpad, scratch_locations, kernel_params->pre_stress_iterations, kernel_params->pre_stress_pattern);
    }
    if (kernel_params->barrier) {
      spin(barrier, blockDim.x * kernel_params->testing_workgroups);
    }

    if (id_1 != id_2) {

      // Thread 0
      test_locations[x_0].store(1, cuda::memory_order_relaxed);
      test_locations[y_0].store(1, thread_0_store);

      // Thread 1
      uint r0 = test_locations[y_1].load(thread_1_load);
      test_locations[z_1].store(1, thread_1_store);

      // Thread 2
      uint r1 = test_locations[z_2].load(thread_2_load);
      uint r2 = test_locations[x_2].load(cuda::memory_order_relaxed);

      cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
      read_results[wg_offset + id_1].r0 = r0;
      read_results[wg_offset + id_2].r1 = r1;
      read_results[wg_offset + id_2].r2 = r2;
    }
  }
  else if (kernel_params->mem_stress) {
    do_stress(scratchpad, scratch_locations, kernel_params->mem_stress_iterations, kernel_params->pre_stress_iterations);
  }
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params) {
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  uint r0 = read_results[id_0].r0;
  uint r1 = read_results[id_0].r1;
  uint r2 = read_results[id_0].r2;

  if (r0 == 1 && r1 == 1 && r2 == 1) {
    test_results->seq0.fetch_add(1);
  }
  else if (r0 == 0 && r1 == 0 && r2 == 0) {
    test_results->seq1.fetch_add(1);
  }
  else if (r0 == 0 && r1 == 0 && r2 == 1) {
    test_results->seq2.fetch_add(1);
  }
  else if (r0 == 1 && r1 == 0 && r2 == 0) {
    test_results->seq3.fetch_add(1);
  }
  else if (r0 == 1 && r1 == 0 && r2 == 1) {
    test_results->interleaved0.fetch_add(1);
  }
  else if (r0 == 1 && r1 == 1 && r2 == 0) {
    test_results->weak.fetch_add(1);
  }
  else {
    test_results->other.fetch_add(1);
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=0, r1=1, r2=1 (seq): " << results->seq0 << "\n";
    std::cout << "r0=0, r1=0, r2=0 (seq): " << results->seq1 << "\n";
    std::cout << "r0=0, r1=0, r2=1 (seq): " << results->seq2 << "\n";
    std::cout << "r0=1, r1=0, r2=0 (seq): " << results->seq3 << "\n";
    std::cout << "r0=1, r1=0, r2=1 (interleaved): " << results->interleaved0 << "\n";
    std::cout << "r0=1, r1=1, r2=0 (weak): " << results->weak << "\n";
    std::cout << "other: " << results->other << "\n";

  }
  return results->weak;
}

