#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cuh"

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  uint* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params) {

  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {
    uint total_ids = blockDim.x * kernel_params->testing_workgroups;
    uint id_0 = shuffled_workgroup * blockDim.x + threadIdx.x;
    uint new_workgroup = stripe_workgroup(shuffled_workgroup, threadIdx.x, kernel_params->testing_workgroups);
    uint id_1 = new_workgroup * blockDim.x + permute_id(threadIdx.x, kernel_params->permute_thread, blockDim.x);
    uint x_0 = id_0 * kernel_params->mem_stride * 2;
    uint y_0 = permute_id(id_0, kernel_params->permute_location, total_ids) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;
    uint x_1 = id_1 * kernel_params->mem_stride * 2;
    uint y_1 = permute_id(id_1, kernel_params->permute_location, total_ids) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;

    if (kernel_params->pre_stress) {
      do_stress(scratchpad, scratch_locations, kernel_params->pre_stress_iterations, kernel_params->pre_stress_pattern);
    }
    if (kernel_params->barrier) {
      spin(barrier, blockDim.x * kernel_params->testing_workgroups);
    }

    test_locations[x_0].store(1, cuda::memory_order_relaxed);
    cuda::atomic_thread_fence(cuda::memory_order_release, cuda::thread_scope_device);
    test_locations[y_0].store(1, cuda::memory_order_relaxed);
    uint r0 = test_locations[y_1].load(cuda::memory_order_relaxed);
    cuda::atomic_thread_fence(cuda::memory_order_acquire, cuda::thread_scope_device);
    uint r1 = test_locations[x_1].load(cuda::memory_order_relaxed);
    cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
    read_results[id_1 * 2 + 1] = r1;
    read_results[id_1 * 2] = r0;
  }
  else if (kernel_params->mem_stress) {
    do_stress(scratchpad, scratch_locations, kernel_params->mem_stress_iterations, kernel_params->pre_stress_iterations);
  }
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  uint* read_results,
  TestResults* test_results,
  KernelParams* kernel_params) {
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  uint x_0 = id_0 * kernel_params->mem_stride * 2;
  uint mem_x_0 = test_locations[x_0];
  uint r0 = read_results[id_0 * 2];
  uint r1 = read_results[id_0 * 2 + 1];
  uint total_ids = blockDim.x * kernel_params->testing_workgroups;
  uint y_0 = permute_id(id_0, kernel_params->permute_location, total_ids) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;
  uint mem_y_0 = test_locations[y_0];

  if ((r0 == 0 && r1 == 0)) {
    test_results->seq0.fetch_add(1);
  }
  else if ((r0 == 1 && r1 == 1)) {
    test_results->seq1.fetch_add(1);
  }
  else if ((r0 == 0 && r1 == 1)) {
    test_results->interleaved0.fetch_add(1);
  }
  else if ((r0 == 1 && r1 == 0)) {
    test_results->weak.fetch_add(1);
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=0, r1=0 (seq): " << results->seq0 << "\n";
    std::cout << "r0=1, r1=1 (seq): " << results->seq1 << "\n";
    std::cout << "r0=0, r1=1 (interleaved): " << results->interleaved0 << "\n";
    std::cout << "r0=1, r1=0 (weak): " << results->weak << "\n";
  }
  return results->weak;
}

