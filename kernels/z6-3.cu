#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cu"

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params,
  TestInstance* test_instances) {

  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

#ifdef ACQ_REL
    cuda::memory_order store_order0 = cuda::memory_order_release;
    cuda::memory_order store_order1 = cuda::memory_order_release;
    cuda::memory_order load_order = cuda::memory_order_acquire;
    #define FENCE0()
    #define FENCE1()
    #define FENCE2()
#elif defined(RELAXED)
    cuda::memory_order store_order0 = cuda::memory_order_relaxed;
    cuda::memory_order store_order1 = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
    #define FENCE0()
    #define FENCE1()
    #define FENCE2()
#elif defined(ALL_FENCE)
    cuda::memory_order store_order0 = cuda::memory_order_relaxed;
    cuda::memory_order store_order1 = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
    #define FENCE0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(FENCE_0)
    cuda::memory_order store_order0 = cuda::memory_order_relaxed;
    cuda::memory_order store_order1 = cuda::memory_order_release;
    cuda::memory_order load_order = cuda::memory_order_acquire;
    #define FENCE0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE1()
    #define FENCE2()
#elif defined(FENCE_1)
    cuda::memory_order store_order0 = cuda::memory_order_release;
    cuda::memory_order store_order1 = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_acquire;
    #define FENCE0()
    #define FENCE1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE2()
#elif defined(FENCE_2)
    cuda::memory_order store_order0 = cuda::memory_order_release;
    cuda::memory_order store_order1 = cuda::memory_order_release;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
    #define FENCE0()
    #define FENCE1()
    #define FENCE2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(FENCE_01)
    cuda::memory_order store_order0 = cuda::memory_order_relaxed;
    cuda::memory_order store_order1 = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_acquire;
    #define FENCE0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE2()
#elif defined(FENCE_02)
    cuda::memory_order store_order0 = cuda::memory_order_relaxed;
    cuda::memory_order store_order1 = cuda::memory_order_release;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
    #define FENCE0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE1()
    #define FENCE2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(FENCE_12)
    cuda::memory_order store_order0 = cuda::memory_order_release;
    cuda::memory_order store_order1 = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
    #define FENCE0()
    #define FENCE1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#else
    cuda::memory_order store_order0 = cuda::memory_order_relaxed;
    cuda::memory_order store_order1 = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
    #define FENCE0()
    #define FENCE1()
    #define FENCE2()
#endif

    // defined for different distributions of threads across threadblocks
    DEFINE_IDS();

    // defined for all three thread three memory locations tests
    THREE_THREAD_THREE_MEM_LOCATIONS();

    PRE_STRESS();

    if (id_0 != id_1 && id_1 != id_2 && id_0 != id_2) {

      // Thread 0
      test_locations[x_0].store(1, cuda::memory_order_relaxed);
      FENCE0()
      test_locations[y_0].store(1, store_order0);

      // Thread 1
      test_locations[y_1].store(2, cuda::memory_order_relaxed);
      FENCE1()
      test_locations[z_1].store(1, store_order1);

      // Thread 2
      uint r0 = test_locations[z_2].load(load_order);
      FENCE2()
      uint r1 = test_locations[x_2].load(cuda::memory_order_relaxed);

      cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
      read_results[wg_offset + id_2].r0 = r0;
      read_results[wg_offset + id_2].r1 = r1;
    }
  }

  MEM_STRESS();
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params,
  bool* weak) {
  RESULT_IDS();
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  uint r0 = read_results[id_0].r0;
  uint r1 = read_results[id_0].r1;
  uint x = test_locations[id_0 * kernel_params->mem_stride * 3];
  uint y_loc = (wg_offset + permute_id(id_0, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 3 + kernel_params->mem_offset;
  uint y = test_locations[y_loc];

  if (x == 0) {
    test_results->na.fetch_add(1); // thread skipped
  }
  else if (y == 2 && r0 == 1 && r1 == 0) {
    test_results->weak.fetch_add(1);
    weak[id_0] = true;
  }
  else {
    test_results->other.fetch_add(1);
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=1, r1=0, y=2 (weak): " << results->weak << "\n";
    std::cout << "thread skipped: " << results->na << "\n";
    std::cout << "other: " << results->other << "\n";
  }
  return results->weak;
}

