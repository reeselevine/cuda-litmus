#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cuh"

__global__ void litmus_test(
    cuda::atomic<uint>* test_locations,
    uint* read_results,
    uint* shuffled_workgroups,
    cuda::atomic<uint>* barrier,
    uint* scratchpad,
    uint* scratch_locations,
    uint* stress_params) {
    uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
    if (shuffled_workgroup < stress_params[9]) {
        uint total_ids = blockDim.x * stress_params[9];
        uint id_0 = shuffled_workgroup * blockDim.x + threadIdx.x;
	uint id_0_final = id_0 % total_ids;
        bool id_0_first_half = id_0 / total_ids == 0;
        uint new_workgroup = stripe_workgroup(shuffled_workgroup, threadIdx.x, stress_params[9]);
        uint id_1 = new_workgroup * blockDim.x + permute_id(threadIdx.x, stress_params[7], blockDim.x);
	uint id_1_final = id_1 % total_ids;
        bool id_1_first_half = id_1 / total_ids == 0;

	uint mem_0;
        if (id_0_first_half) {
            mem_0 = id_0_final * stress_params[10] * 2;
        } else {
            mem_0 = permute_id(id_0_final, stress_params[8], total_ids) * stress_params[10] * 2 + stress_params[11];
        }
        uint x_1 = (id_1_final) * stress_params[10] * 2;
        uint y_1 = (permute_id(id_1_final, stress_params[8], total_ids)) * stress_params[10] * 2 + stress_params[11];

        if (stress_params[4]) {
            do_stress(scratchpad, scratch_locations, stress_params[5], stress_params[6]);
        }
        if (stress_params[0]) {
            spin(barrier, blockDim.x * stress_params[9]);
        }

	test_locations[mem_0].store(1, cuda::memory_order_relaxed); // write to either x or y depending on thread

	if (id_0_final != id_1_final) {
            if (id_1_first_half) { // one observer thread reads x then y
		uint r0 = test_locations[x_1].load(cuda::memory_order_relaxed);
		uint r1 = test_locations[y_1].load(cuda::memory_order_relaxed);
	        cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
	        read_results[id_1_final * 4] = r0;
	        read_results[id_1_final * 4 + 1] = r1;
            } else { // other observer thread reads y then x
		uint r2 = test_locations[y_1].load(cuda::memory_order_relaxed);
		uint r3 = test_locations[x_1].load(cuda::memory_order_relaxed);
	        cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
	        read_results[id_1_final * 4 + 2] = r2;
	        read_results[id_1_final * 4 + 3] = r3;
            }
        }
    } else if (stress_params[1]) {
        do_stress(scratchpad, scratch_locations, stress_params[2], stress_params[3]);
    }
}

__global__ void check_results (
    cuda::atomic<uint>* test_locations,
    uint* read_results,
    TestResults* test_results,
    uint* stress_params) {
    uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
    if (id_0 < (blockDim.x * stress_params[9])/2) {
        uint x_0 = id_0 * stress_params[10] * 2;
	uint mem_x_0 = test_locations[x_0];
        uint r0 = read_results[id_0 * 4];
        uint r1 = read_results[id_0 * 4 + 1];
        uint r2 = read_results[id_0 * 4 + 2];
        uint r3 = read_results[id_0 * 4 + 3];

        if (r0 == 0 && r1 == 0 && r2 == 0 && r3 == 0) { // both observers run first
	    test_results->seq0.fetch_add(1);
        } else if (r0 == 1 && r1 == 1 && r2 == 1 && r3 == 1) { // both observers run last
	    test_results->seq1.fetch_add(1);
        } else if (r0 == 0 && r1 == 0 && r2 == 1 && r3 == 1) { // first observer runs first
	    test_results->seq2.fetch_add(1);
        } else if (r0 == 1 && r1 == 1 && r2 == 0 && r3 == 0) { // second observer runs first
	    test_results->seq3.fetch_add(1);
        } else if (r0 == r1 && r2 != r3) { // second observer interleaved
	    test_results->seq_inter0.fetch_add(1);
        } else if (r0 != r1 && r2 == r3) { // first observer interleaved
	    test_results->seq_inter1.fetch_add(1);
        } else if (r0 == 0 && r1 == 1 && r2 == 0 && r3 == 1) { // both interleaved
	    test_results->interleaved0.fetch_add(1);
        } else if (r0 == 0 && r1 == 1 && r2 == 1 && r3 == 0) { // both interleaved
	    test_results->interleaved1.fetch_add(1);
        } else if (r0 == 1 && r1 == 0 && r2 == 0 && r3 == 1) { // both interleaved
	    test_results->interleaved2.fetch_add(1);
        } else if (r0 == 1 && r1 == 0 && r2 == 1 && r3 == 0) { // observer threads see x/y in different orders
	    test_results->weak.fetch_add(1);
        } else {
	    test_results->other.fetch_add(1);
        }
    }
}

int host_check_results(TestResults* results, bool print) {
    if (print) {

        std::cout << "r0=0, r1=0, r2=0, r3=0 (seq): " << results->seq0 << "\n";
        std::cout << "r0=1, r1=1, r2=1, r3=1 (seq): " << results->seq1 << "\n";
	std::cout << "r0=0, r1=0, r2=1, r3=1 (seq): " << results->seq2 << "\n";
	std::cout << "r0=1, r1=1, r2=0, r3=0 (seq): " << results->seq3 << "\n";
	std::cout << "r0 == r1, r2 != r3 (seq/interleaved): " << results->seq_inter0 << "\n";
	std::cout << "r0 != r1, r2 == r3 (interleaved/seq): " << results->seq_inter1 << "\n";
	std::cout << "r0=0, r1=1, r2=0, r3=1 (interleaved): " << results->interleaved0 << "\n";
	std::cout << "r0=0, r1=1, r2=1, r3=0 (interleaved): " << results->interleaved1 << "\n";
	std::cout << "r0=1, r1=0, r2=0, r3=1 (interleaved): " << results->interleaved2 << "\n";
	std::cout << "r0=1, r1=0, r2=1, r3=0 (weak): " << results->weak << "\n";
	std::cout << "other: " << results->other << "\n";
    }
    return results->weak;
}

