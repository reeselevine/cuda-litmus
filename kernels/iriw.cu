#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cuh"

#ifdef TB_0_1_2_3
#include "tb-0-1-2-3.h"
#elif defined(TB_01_23)
#include "tb-01-23.h"
#elif defined(TB_0123)
#include "tb-0123.h"
#else
#include "tb-0-1-2-3.h" // default to all different threadblocks
#endif

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params) {
  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

#ifdef ACQUIRE
    cuda::memory_order first_mem_order = cuda::memory_order_acquire;
    #define FENCE()
#elif defined(ACQ_FENCE_BLOCK)
    cuda::memory_order first_mem_order = cuda::memory_order_relaxed;
    #define FENCE() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, cuda::thread_scope_block);
#elif defined(ACQ_FENCE_DEVICE)
    cuda::memory_order first_mem_order = cuda::memory_order_relaxed;
    #define FENCE() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, cuda::thread_scope_device);
#elif defined(ACQ_FENCE_SYSTEM)
    cuda::memory_order first_mem_order = cuda::memory_order_relaxed;
    #define FENCE() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, cuda::thread_scope_system);
#elif defined(RELAXED)
    cuda::memory_order first_mem_order = cuda::memory_order_relaxed;
    #define FENCE()
#else
    cuda::memory_order first_mem_order = cuda::memory_order_relaxed; // default to relaxed
    #define FENCE()
#endif

    // defined for different distributions of threads across threadblocks
    DEFINE_IDS();

    uint mem_0;
    if (id_0_first_half) {
      mem_0 = (wg_offset + id_0_final) * kernel_params->mem_stride * 2;
    }
    else {
      mem_0 = (wg_offset + permute_id(id_0_final, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;
    }
    uint x_1 = (wg_offset + id_1_final) * kernel_params->mem_stride * 2;
    uint y_1 = (wg_offset + permute_id(id_1_final, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;

    PRE_STRESS();

    if (id_0_final != id_1_final) {

      test_locations[mem_0].store(1, cuda::memory_order_relaxed); // write to either x or y depending on thread

      if (id_1_first_half) { // one observer thread reads x then y
        uint r0 = test_locations[x_1].load(first_mem_order);
        FENCE()
        uint r1 = test_locations[y_1].load(cuda::memory_order_relaxed);
        cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
        read_results[wg_offset + id_1_final].r0 = r0;
        read_results[wg_offset + id_1_final].r1 = r1;
      }
      else { // other observer thread reads y then x
        uint r2 = test_locations[y_1].load(first_mem_order);
        FENCE()
        uint r3 = test_locations[x_1].load(cuda::memory_order_relaxed);
        cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
        read_results[wg_offset + id_1_final].r2 = r2;
        read_results[wg_offset + id_1_final].r3 = r3;
      }
    }
  }
  MEM_STRESS();
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params) {
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  if (id_0 < (blockDim.x * kernel_params->testing_workgroups) / 2) {
    uint x = test_locations[id_0 * kernel_params->mem_stride * 2];
    uint r0 = read_results[id_0].r0;
    uint r1 = read_results[id_0].r1;
    uint r2 = read_results[id_0].r2;
    uint r3 = read_results[id_0].r3;

    if (x == 0) {
      test_results->na.fetch_add(1); // thread skipped
    }
    else if (r0 == 0 && r1 == 0 && r2 == 0 && r3 == 0) { // both observers run first
      test_results->res0.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 1 && r2 == 1 && r3 == 1) { // both observers run last
      test_results->res1.fetch_add(1);
    }
    else if (r0 == 0 && r1 == 0 && r2 == 1 && r3 == 1) { // first observer runs first
      test_results->res2.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 1 && r2 == 0 && r3 == 0) { // second observer runs first
      test_results->res3.fetch_add(1);
    }
    else if (r0 == r1 && r2 != r3) { // second observer interleaved
      test_results->res4.fetch_add(1);
    }
    else if (r0 != r1 && r2 == r3) { // first observer interleaved
      test_results->res5.fetch_add(1);
    }
    else if (r0 == 0 && r1 == 1 && r2 == 0 && r3 == 1) { // both interleaved
      test_results->res6.fetch_add(1);
    }
    else if (r0 == 0 && r1 == 1 && r2 == 1 && r3 == 0) { // both interleaved
      test_results->res7.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 0 && r2 == 0 && r3 == 1) { // both interleaved
      test_results->res8.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 0 && r2 == 1 && r3 == 0) { // observer threads see x/y in different orders
      test_results->weak.fetch_add(1);
    }
    else {
      test_results->other.fetch_add(1);
    }
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=0, r1=0, r2=0, r3=0 (seq): " << results->res0 << "\n";
    std::cout << "r0=1, r1=1, r2=1, r3=1 (seq): " << results->res1 << "\n";
    std::cout << "r0=0, r1=0, r2=1, r3=1 (seq): " << results->res2 << "\n";
    std::cout << "r0=1, r1=1, r2=0, r3=0 (seq): " << results->res3 << "\n";
    std::cout << "r0 == r1, r2 != r3 (seq/interleaved): " << results->res4 << "\n";
    std::cout << "r0 != r1, r2 == r3 (interleaved/seq): " << results->res5 << "\n";
    std::cout << "r0=0, r1=1, r2=0, r3=1 (interleaved): " << results->res6 << "\n";
    std::cout << "r0=0, r1=1, r2=1, r3=0 (interleaved): " << results->res7 << "\n";
    std::cout << "r0=1, r1=0, r2=0, r3=1 (interleaved): " << results->res8 << "\n";
    std::cout << "r0=1, r1=0, r2=1, r3=0 (weak): " << results->weak << "\n";
    std::cout << "thread skipped: " << results->na << "\n";
    std::cout << "other: " << results->other << "\n";
  }
  return results->weak;
}

