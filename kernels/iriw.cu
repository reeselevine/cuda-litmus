#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cuh"

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params) {
  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

#ifdef ACQUIRE
    cuda::memory_order thread_1_order = cuda::memory_order_acquire;
    cuda::memory_order thread_3_order = cuda::memory_order_acquire;
    #define FENCE_1()
    #define FENCE_3()
#elif defined(THREAD_1_FENCE)
    cuda::memory_order thread_1_order = cuda::memory_order_relaxed;
    cuda::memory_order thread_3_order = cuda::memory_order_acquire;
    #define FENCE_1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_3()
#elif defined(THREAD_3_FENCE)
    cuda::memory_order thread_1_order = cuda::memory_order_acquire;
    cuda::memory_order thread_3_order = cuda::memory_order_relaxed;
    #define FENCE_1()
    #define FENCE_3() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(BOTH_FENCE)
    cuda::memory_order thread_1_order = cuda::memory_order_relaxed;
    cuda::memory_order thread_3_order = cuda::memory_order_relaxed;
    #define FENCE_1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_3() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(RELAXED)
    cuda::memory_order thread_1_order = cuda::memory_order_relaxed;
    cuda::memory_order thread_3_order = cuda::memory_order_relaxed;
    #define FENCE_1()
    #define FENCE_3()
#else
    cuda::memory_order thread_1_order = cuda::memory_order_relaxed;
    cuda::memory_order thread_3_order = cuda::memory_order_relaxed;
    #define FENCE_1()
    #define FENCE_3()
#endif

    // defined for different distributions of threads across threadblocks
    DEFINE_IDS();

    uint mem_0;
    if (id_0_first_half) {
      mem_0 = (wg_offset + id_0_final) * kernel_params->mem_stride * 2;
    }
    else {
      mem_0 = (wg_offset + permute_id(id_0_final, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;
    }
    uint x_1 = (wg_offset + id_1_final) * kernel_params->mem_stride * 2;
    uint y_1 = (wg_offset + permute_id(id_1_final, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 2 + kernel_params->mem_offset;

    PRE_STRESS();

    if (id_0_final != id_1_final) {

      test_locations[mem_0].store(1, cuda::memory_order_relaxed); // write to either x or y depending on thread

      if (id_1_first_half) { // one observer thread reads x then y
        uint r0 = test_locations[x_1].load(thread_1_order);
        FENCE_1()
        uint r1 = test_locations[y_1].load(cuda::memory_order_relaxed);
        cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
        read_results[wg_offset + id_1_final].r0 = r0;
        read_results[wg_offset + id_1_final].r1 = r1;
      }
      else { // other observer thread reads y then x
        uint r2 = test_locations[y_1].load(thread_3_order);
        FENCE_3()
        uint r3 = test_locations[x_1].load(cuda::memory_order_relaxed);
        cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
        read_results[wg_offset + id_1_final].r2 = r2;
        read_results[wg_offset + id_1_final].r3 = r3;
      }
    }
  }
  MEM_STRESS();
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params) {
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  if (id_0 < (blockDim.x * kernel_params->testing_workgroups) / 2) {
    uint x = test_locations[id_0 * kernel_params->mem_stride * 2];
    uint r0 = read_results[id_0].r0;
    uint r1 = read_results[id_0].r1;
    uint r2 = read_results[id_0].r2;
    uint r3 = read_results[id_0].r3;

    if (x == 0) {
      test_results->na.fetch_add(1); // thread skipped
    }
    else if (r0 == 0 && r1 == 0 && r2 == 0 && r3 == 0) { // both observers run first
      test_results->res0.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 1 && r2 == 1 && r3 == 1) { // both observers run last
      test_results->res1.fetch_add(1);
    }
    else if (r0 == 0 && r1 == 0 && r2 == 1 && r3 == 1) { // first observer runs first
      test_results->res2.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 1 && r2 == 0 && r3 == 0) { // second observer runs first
      test_results->res3.fetch_add(1);
    }
    else if (r0 == r1 && r2 != r3) { // second observer interleaved
      test_results->res4.fetch_add(1);
    }
    else if (r0 != r1 && r2 == r3) { // first observer interleaved
      test_results->res5.fetch_add(1);
    }
    else if (r0 == 0 && r1 == 1 && r2 == 0 && r3 == 1) { // both interleaved
      test_results->res6.fetch_add(1);
    }
    else if (r0 == 0 && r1 == 1 && r2 == 1 && r3 == 0) { // both interleaved
      test_results->res7.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 0 && r2 == 0 && r3 == 1) { // both interleaved
      test_results->res8.fetch_add(1);
    }
    else if (r0 == 1 && r1 == 0 && r2 == 1 && r3 == 0) { // observer threads see x/y in different orders
      test_results->weak.fetch_add(1);
    }
    else {
      test_results->other.fetch_add(1);
    }
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=0, r1=0, r2=0, r3=0 (seq): " << results->res0 << "\n";
    std::cout << "r0=1, r1=1, r2=1, r3=1 (seq): " << results->res1 << "\n";
    std::cout << "r0=0, r1=0, r2=1, r3=1 (seq): " << results->res2 << "\n";
    std::cout << "r0=1, r1=1, r2=0, r3=0 (seq): " << results->res3 << "\n";
    std::cout << "r0 == r1, r2 != r3 (seq/interleaved): " << results->res4 << "\n";
    std::cout << "r0 != r1, r2 == r3 (interleaved/seq): " << results->res5 << "\n";
    std::cout << "r0=0, r1=1, r2=0, r3=1 (interleaved): " << results->res6 << "\n";
    std::cout << "r0=0, r1=1, r2=1, r3=0 (interleaved): " << results->res7 << "\n";
    std::cout << "r0=1, r1=0, r2=0, r3=1 (interleaved): " << results->res8 << "\n";
    std::cout << "r0=1, r1=0, r2=1, r3=0 (weak): " << results->weak << "\n";
    std::cout << "thread skipped: " << results->na << "\n";
    std::cout << "other: " << results->other << "\n";
  }
  return results->weak;
}

