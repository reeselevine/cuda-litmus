#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cu"

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params,
  TestInstance* test_instances) {
  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

    uint total_ids = blockDim.x * kernel_params->testing_workgroups;
    uint id_0 = shuffled_workgroup * blockDim.x + threadIdx.x;
    uint workgroup_1 = stripe_workgroup(shuffled_workgroup, threadIdx.x, kernel_params->testing_workgroups);
    uint id_1 = workgroup_1 * blockDim.x + threadIdx.x;

    uint x_0 = id_0 * kernel_params->mem_stride * 2;
    uint permute_id_0 = permute_id(id_0, kernel_params->permute_location, total_ids);
    uint y_0 = permute_id_0 * kernel_params->mem_stride * 2 + kernel_params->mem_offset;

    uint x_1 = id_1 * kernel_params->mem_stride * 2;
    uint permute_id_1 = permute_id(id_1, kernel_params->permute_location, total_ids);
    uint y_1 = permute_id_1 * kernel_params->mem_stride * 2 + kernel_params->mem_offset;

    PRE_STRESS();

    if (id_0 != id_1) {

      test_locations[x_0].store(1, cuda::memory_order_relaxed);
      test_locations[y_0].store(1, cuda::memory_order_release);

      test_locations[y_1].store(2, cuda::memory_order_relaxed);
      uint r0 = test_locations[x_1].load(cuda::memory_order_relaxed);

      cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
      read_results[id_1].r0 = r0;
    }
  }
  MEM_STRESS();
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params,
  bool* weak) {
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  uint x = test_locations[id_0 * kernel_params->mem_stride * 2];
  uint permute_id_0 = permute_id(id_0, kernel_params->permute_location, gridDim.x * blockDim.x);
  uint y = test_locations[permute_id_0 * kernel_params->mem_stride * 2 + kernel_params->mem_offset];
  uint r0 = read_results[id_0].r0;

  if (x == 0) {
    test_results->na.fetch_add(1); // thread skipped
  }
  else if (r0 == 0 && y == 2) { // weak behavior
    test_results->weak.fetch_add(1);
  }
  else {
    test_results->other.fetch_add(1);
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=0, y=2 (weak): " << results->weak << "\n";
    std::cout << "other: " << results->other << "\n";
    std::cout << "thread skipped: " << results->na << "\n";
  }
  return results->weak;
}

